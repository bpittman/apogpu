 
#include <hip/hip_runtime.h>
#include "stdio.h"

// printf() is only supported
// for devices of compute capability 2.0 and higher

__global__ void gainKernel(float* data_d) {
   float gain = 0.5f;
   unsigned int idx = blockIdx.x*32 + threadIdx.x;
   data_d[idx] = data_d[idx] * gain;
   return;
}

void gpusetup(float *data, int channels, int samples) {
   float *data_d = NULL;

   printf("frames: %d\n",samples);

   // Allocate device memory and Transfer host arrays M and N
   hipMalloc(&data_d, sizeof(float)*samples);

   hipMemcpy(data_d, data, sizeof(float)*samples, hipMemcpyHostToDevice);
   //cudaMemcpy(data, data_d, samples, cudaMemcpyHostToDevice);

   // Stage A:  Setup the kernel execution configuration parameters
   dim3 dimGrid(samples/32,1,1);
   dim3 dimBlock(32,1,1);

   printf("gpusetup: %f\n",data[0]);

   // Stage B: Launch the kernel!! -- using the appropriate function arguments
   gainKernel<<<dimGrid, dimBlock>>>(data_d);

   hipMemcpy(data, data_d, sizeof(float)*samples, hipMemcpyDeviceToHost);
   //cudaMemcpy(data_d, data, samples, cudaMemcpyDeviceToHost);

   printf("gpusetup: %f\n",data[0]);
   if(hipGetLastError() != hipSuccess) { printf("error!\n"); }

   // End of solution Part 3 ============================================


   // Free device matrices
   hipFree(data_d);
}
