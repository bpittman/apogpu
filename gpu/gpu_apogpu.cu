#include "hip/hip_runtime.h"
#include <stdio.h>
#include "apogpu.h"

__global__ void gainKernel(float* data_d) {
   float gain = 0.5f;
   unsigned int idx = blockIdx.x*BLOCK_SIZE + threadIdx.x;
   data_d[idx] = data_d[idx] * gain;
   return;
}

__global__ void delayKernel(float* data_d, int channels, int samples, float decay, int delay_length, int base) {
   unsigned int idx = base + blockIdx.x*BLOCK_SIZE + threadIdx.x;
   if(idx >= samples) return;
   data_d[idx+(delay_length*channels)] += data_d[idx]*decay;
   return;
}

void cudasafe( hipError_t error, char* message) {
   if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %s\n",message,hipGetErrorString(error)); exit(-1); }
}

void launchGainKernel(float* data_d, int samples) {
   // Stage A:  Setup the kernel execution configuration parameters
   dim3 dimGrid(samples/BLOCK_SIZE,1,1);
   dim3 dimBlock(BLOCK_SIZE,1,1);

   // Stage B: Launch the kernel!! -- using the appropriate function arguments
   gainKernel<<<dimGrid, dimBlock>>>(data_d);


   return;
}

void launchDelayKernel(float* data_d, int channels, int samples, float decay, int delay_length) {
   int i;
   for(i=0;i<samples;i+=delay_length) {
      // Stage A:  Setup the kernel execution configuration parameters
      dim3 dimGrid(delay_length/BLOCK_SIZE,1,1);
      dim3 dimBlock(BLOCK_SIZE,1,1);

      // Stage B: Launch the kernel!! -- using the appropriate function arguments
      delayKernel<<<dimGrid, dimBlock>>>(data_d, channels, samples, decay, delay_length, i);
   }

   return;
}

void gpusetup(float *data, int channels, int sample_rate, int samples) {
   float *data_d = NULL;
   float time;
   hipEvent_t start, stop;

   printf("frames: %d\n",samples);

   cudasafe(hipEventCreate(&start),"hipEventCreate");
   cudasafe(hipEventCreate(&stop),"hipEventCreate");
   cudasafe(hipEventRecord(start, 0),"hipEventRecord");

   // Allocate device memory and Transfer host arrays M and N
   cudasafe(hipMalloc(&data_d, sizeof(float)*samples),"hipMalloc");

   printf("gpusetup: %f\n",data[0]);

   cudasafe(hipMemcpy(data_d, data, sizeof(float)*samples, hipMemcpyHostToDevice),"cudaMempy");

   //launchGainKernel(data_d, samples);
   launchDelayKernel(data_d, channels, samples, 0.5f, (int)200*(sample_rate/1000));

   cudasafe(hipMemcpy(data, data_d, sizeof(float)*samples, hipMemcpyDeviceToHost),"hipMemcpy");

   cudasafe(hipEventRecord(stop, 0),"hipEventRecord");
   cudasafe(hipEventSynchronize(stop),"hipEventSynchronize");
   cudasafe(hipEventElapsedTime(&time, start, stop),"cudaEvenElapsedTime");

   printf("gpusetup: %f\n",data[0]);

   printf("Time to generate (gpu):  %f ms \n", time);

   int chan,k;
   int delay_length = 200*(sample_rate/1000);
   int globalcount=0;
   float decay  = 0.5;

   cudasafe(hipEventCreate(&start),"hipEventCreate");
   cudasafe(hipEventCreate(&stop),"hipEventCreate");
   cudasafe(hipEventRecord(start, 0),"hipEventRecord");

   for (chan = 0 ; chan < channels ; chan ++) {
      for (k = chan ; k+(delay_length*channels) < samples; k+= channels) {
         data[k+(delay_length*channels)] += data[k]*decay;
         globalcount++;
      }
   }

   cudasafe(hipEventRecord(stop, 0),"hipEventRecord");
   cudasafe(hipEventSynchronize(stop),"hipEventSynchronize");
   cudasafe(hipEventElapsedTime(&time, start, stop),"cudaEvenElapsedTime");

   printf("Time to generate (cpu):  %f ms \n", time);

   // End of solution Part 3 ============================================


   // Free device matrices
   cudasafe(hipFree(data_d),"hipFree");
}
