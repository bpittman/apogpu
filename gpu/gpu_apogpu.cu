#include "hip/hip_runtime.h"
#include <stdio.h>
#include "apogpu.h"

__device__ __constant__ int d_delay_length_x_channels;
__device__ __constant__ float d_decay;
__device__ __constant__ int d_samples;

__global__ void gainKernel(float* data_d) {
   float gain = 0.5f;
   unsigned int idx = blockIdx.x*BLOCK_SIZE + threadIdx.x;
   data_d[idx] = data_d[idx] * gain;
   return;
}

__global__ void lowPassKernel(float* data_d, float* results_d, int channels) {
   extern __shared__ float data_s[];
   float h = 0.03125f;
   unsigned int tidx = threadIdx.x;
   unsigned int idx = blockIdx.x*BLOCK_SIZE + tidx;

   //load the last 256 frames into shared memory
   for(int i=0;i<channels;++i) {
      data_s[(tidx+32)*channels+i] = data_d[(idx*channels)+i];
   }

   if(idx<32*channels) {
      __syncthreads();
      return;
   }

   //load the first 32 frames into shared memory
   if(tidx<32) {
      for(int i=0;i<channels;++i) {
         data_s[(tidx*channels)+i] = data_d[((idx-32)*channels)+i];
      }
   }

   __syncthreads();

   float x = 0;
   for(int i=0;i<32*channels;i+=channels) {
      x += data_s[tidx-i+(32*channels)]*h;
   }
   results_d[idx] = x;
   return;
}

__global__ void delayKernel(float* data_d, int base) {
   unsigned int idx = base + blockIdx.x*BLOCK_SIZE + threadIdx.x;
   if(idx >= d_samples) return;
   data_d[idx+(d_delay_length_x_channels)] += data_d[idx]*d_decay;
   return;
}

void cudasafe( hipError_t error, char* message) {
   if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %s\n",message,hipGetErrorString(error)); exit(-1); }
}

void launchGainKernel(float* data_d, int samples) {
   // Stage A:  Setup the kernel execution configuration parameters
   dim3 dimGrid(samples/BLOCK_SIZE,1,1);
   dim3 dimBlock(BLOCK_SIZE,1,1);

   // Stage B: Launch the kernel!! -- using the appropriate function arguments
   gainKernel<<<dimGrid, dimBlock>>>(data_d);

   hipDeviceSynchronize();
   hipError_t error = hipGetLastError();
   if(error != hipSuccess) {
      printf("gainKernel error: %s\n", hipGetErrorString(error));
   }

   return;
}

void launchLowPassKernel(float* data_d, float* results_d, int samples, int channels) {
   // Stage A:  Setup the kernel execution configuration parameters
   dim3 dimGrid(samples/BLOCK_SIZE,1,1);
   dim3 dimBlock(BLOCK_SIZE,1,1);
   int shared_size = ((BLOCK_SIZE+32)*channels)*sizeof(float);

   // Stage B: Launch the kernel!! -- using the appropriate function arguments
   lowPassKernel<<<dimGrid, dimBlock, shared_size>>>(data_d, results_d, channels);

   hipDeviceSynchronize();
   hipError_t error = hipGetLastError();
   if(error != hipSuccess) {
      printf("lowPassKernel error: %s\n", hipGetErrorString(error));
   }

   return;
}

void launchDelayKernel(float* data_d, int channels, int samples, float decay, int delay_length) {
   int i;
   int delay_length_x_channels = delay_length*channels;
   cudasafe(hipMemcpyToSymbol(HIP_SYMBOL("d_delay_length_x_channels"),&delay_length_x_channels, sizeof(int)),"hipMemcpyToSymbol");
   cudasafe(hipMemcpyToSymbol(HIP_SYMBOL("d_decay"),&decay,sizeof(float)),"hipMemcpyToSymbol");
   cudasafe(hipMemcpyToSymbol(HIP_SYMBOL("d_samples"),&samples,sizeof(int)),"hipMemcpyToSymbol");
   
   for(i=0;i<samples;i+=delay_length_x_channels) {
      // Stage A:  Setup the kernel execution configuration parameters
      dim3 dimGrid(delay_length_x_channels/BLOCK_SIZE,1,1);
      dim3 dimBlock(BLOCK_SIZE,1,1);

      // Stage B: Launch the kernel!! -- using the appropriate function arguments
      delayKernel<<<dimGrid, dimBlock>>>(data_d, i);

      hipDeviceSynchronize();
      hipError_t error = hipGetLastError();
      if(error != hipSuccess) {
         printf("delayKernel error: %s\n", hipGetErrorString(error));
      }
   }

   return;
}

void gpusetup(float *data, int channels, int sample_rate, int samples) {
   float *data_d = NULL, *results_d = NULL;
   float time;
   hipEvent_t start, stop;

   printf("frames: %d\n",samples);

   cudasafe(hipEventCreate(&start),"hipEventCreate");
   cudasafe(hipEventCreate(&stop),"hipEventCreate");
   cudasafe(hipEventRecord(start, 0),"hipEventRecord");

   cudasafe(hipMalloc(&data_d, sizeof(float)*samples),"hipMalloc");
   cudasafe(hipMalloc(&results_d, sizeof(float)*samples),"hipMalloc");

   printf("gpusetup: %f\n",data[0]);

   cudasafe(hipMemcpy(data_d, data, sizeof(float)*samples, hipMemcpyHostToDevice),"cudaMempy");

   launchGainKernel(data_d, samples);
   launchDelayKernel(data_d, channels, samples, 0.5f, (int)256*(sample_rate/1000));
   launchLowPassKernel(data_d, results_d, samples, channels);

   //cudasafe(hipMemcpy(data, data_d, sizeof(float)*samples, hipMemcpyDeviceToHost),"hipMemcpy");
   cudasafe(hipMemcpy(data, results_d, sizeof(float)*samples, hipMemcpyDeviceToHost),"hipMemcpy");

   cudasafe(hipEventRecord(stop, 0),"hipEventRecord");
   cudasafe(hipEventSynchronize(stop),"hipEventSynchronize");
   cudasafe(hipEventElapsedTime(&time, start, stop),"cudaEvenElapsedTime");

   printf("gpusetup: %f\n",data[0]);

   printf("Time to generate (gpu):  %f ms \n", time);

   int chan,k,i;
   int delay_length = 256*(sample_rate/1000);
   int globalcount=0;
   float decay  = 0.5;
   float h = 0.03125f;
   float *results=NULL;
   float channel_gain = 0.5f;

   results = (float*)malloc(sizeof(float)*samples);
   if(results == NULL)
   {
      printf("malloc failed!\n");
      return;
   }

   cudasafe(hipEventCreate(&start),"hipEventCreate");
   cudasafe(hipEventCreate(&stop),"hipEventCreate");
   cudasafe(hipEventRecord(start, 0),"hipEventRecord");

   for (chan = 0 ; chan < channels ; chan ++) {
      for (k = chan ; k+(delay_length*channels) < samples; k+= channels) {
         data[k] *= channel_gain;
      }
   }

   for (chan = 0 ; chan < channels ; chan ++) {
      for (k = chan ; k+(delay_length*channels) < samples; k+= channels) {
         data[k+(delay_length*channels)] += data[k]*decay;
         globalcount++;
      }
   }

    for(k=0;k<samples;++k) {
        if(k<32*channels) continue;
        float x = 0;
        for(i=0; i<32*channels; i+=channels) {
            x += data[(k-i)]*h;
        }
        results[k] = x;
    }


   cudasafe(hipEventRecord(stop, 0),"hipEventRecord");
   cudasafe(hipEventSynchronize(stop),"hipEventSynchronize");
   cudasafe(hipEventElapsedTime(&time, start, stop),"cudaEvenElapsedTime");

   printf("Time to generate (cpu):  %f ms \n", time);

   // End of solution Part 3 ============================================


   // Free device matrices
   cudasafe(hipFree(data_d),"hipFree");
}
