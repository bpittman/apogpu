
#include <hip/hip_runtime.h>
__global__ void gainKernel(int* data) {
   return;
}

void gpusetup(int *data, int channels, int samples) {
   int *data_d;

   // Allocate device memory and Transfer host arrays M and N
   hipMalloc((void**) &data_d, channels*samples);

   hipMemcpy(data_d, data, channels*samples, hipMemcpyHostToDevice);

   // Stage A:  Setup the kernel execution configuration parameters
   dim3 dimGrid(1,1,1);
   dim3 dimBlock(1,1,1);
   // Stage B: Launch the kernel!! -- using the appropriate function arguments
   gainKernel<<<dimGrid, dimBlock>>>(data_d);

   hipMemcpy(data_d, data, channels*samples, hipMemcpyDeviceToHost);

   // End of solution Part 3 ============================================


   // Free device matrices
   hipFree(data_d);
}
